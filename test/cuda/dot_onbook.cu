#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../book.h"
#include <iostream>
#include <iomanip>

using std::cout;
using std::endl;
using std::ios;

#define imin(a,b) (a<b?a:b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid =
            imin( 32, (N+threadsPerBlock-1) / threadsPerBlock );


__global__ void dot( int *a, int *b, int *c ) {
    __shared__ int cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    int   temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    // set the cache values
    cache[cacheIndex] = temp;
    
    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}


int DotProductOnBook( void ) {
    int   *a, *b, c, *partial_c;
    int   *dev_a, *dev_b, *dev_partial_c;

    // allocate memory on the cpu side
    a = (int*)malloc( N*sizeof(int) );
    b = (int*)malloc( N*sizeof(int) );
    partial_c = (int*)malloc( blocksPerGrid*sizeof(int) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              N*sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b,
                              N*sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_partial_c,
                              blocksPerGrid*sizeof(int) ) );

    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = 1;
        b[i] = 1;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N*sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N*sizeof(int),
                              hipMemcpyHostToDevice ) ); 

    dot<<<blocksPerGrid,threadsPerBlock>>>( dev_a, dev_b,
                                            dev_partial_c );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( partial_c, dev_partial_c,
                              blocksPerGrid*sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // finish up on the CPU side
    c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    }

    #define sum_squares(x)  (x*(x+1)*(2*x+1)/6)
    cout.setf(ios::fixed);
    cout << "reuslt on GPU is " << c << endl
         << "result on CPU is " << sum_squares( (int)(N - 1) ) << endl
         << "The difference is " << c - sum_squares((int)N - 1) << endl;

    // free memory on the gpu side
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_partial_c ) );

    // free memory on the cpu side
    free( a );
    free( b );
    free( partial_c );
}
