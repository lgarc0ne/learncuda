#include "hip/hip_runtime.h"
#include "tools.hpp"
#include "../cpu_anim.h"
#include "../book.h"

#include <iostream>
#include <iomanip>

using std::cout;
using std::endl;
using std::ios;


__global__ void copy_const_kernel(float *iptr, const float *cptr) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (cptr[offset] != 0) iptr[offset] = cptr[offset];
}

__global__ void blend_kernel(float *outSrc, float *inSrc) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    int left = offset - 1;
    int right = offset + 1;
    if (x == 0) ++left;
    if (x == DIM - 1) --right;

    int up = offset - DIM;
    int down = offset + DIM;
    if (y == 0) up += DIM;
    if (y == DIM - 1) down -= DIM;


    outSrc[offset] = inSrc[offset] + SPEED * (inSrc[up] + inSrc[down] +
                                              inSrc[left] + inSrc[right] -
                                              inSrc[offset] * 4);
}


static void AnimGPU(DataBlock *d, int  ticks) {
    CUDA_CHECK_ERROR(hipEventRecord(d->start, 0));
    dim3 grids(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    CPUAnimBitmap *bitmap = d->bitmap;

    for (int i = 0; i < 99; ++i) {
        copy_const_kernel<<<grids, threads>>>(d->dev_inSrc, d->dev_constSrc);
        blend_kernel<<<grids, threads>>>(d->dev_outSrc, d->dev_inSrc);

       std::swap(d->dev_inSrc, d->dev_outSrc);
    }
    float_to_color<<<grids, threads>>>(d->outputBitmap, d->dev_inSrc);

    CUDA_CHECK_ERROR(hipMemcpy(bitmap->get_ptr(), d->outputBitmap,
                                bitmap->image_size(), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipEventRecord(d->stop, 0));
    CUDA_CHECK_ERROR(hipEventSynchronize(d->stop));
    float elapsedTime;
    CUDA_CHECK_ERROR(hipEventElapsedTime(&elapsedTime, d->start, d->stop));
    d->totalTime += elapsedTime;
    ++d->frame;
    cout.setf(ios::fixed);
    cout << "Average Time per frame: " << std::setw(4)
         << d->totalTime / d->frame << endl;
}

static void AnimExit(DataBlock *d) {
    CUDA_CHECK_ERROR(hipFree(d->dev_constSrc));
    CUDA_CHECK_ERROR(hipFree(d->dev_inSrc));
    CUDA_CHECK_ERROR(hipFree(d->dev_outSrc));

    CUDA_CHECK_ERROR(hipEventDestroy(d->start));
    CUDA_CHECK_ERROR(hipEventDestroy(d->stop));
}


void NaiveThermalConductionAnim() {
    DataBlock data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frame = 0;
    CUDA_CHECK_ERROR(hipEventCreate(&data.start));
    CUDA_CHECK_ERROR(hipEventCreate(&data.stop));

    CUDA_CHECK_ERROR(hipMalloc((void**)&data.outputBitmap, bitmap.image_size()));

    CUDA_CHECK_ERROR(hipMalloc((void**)&data.dev_inSrc, bitmap.image_size()));
    CUDA_CHECK_ERROR(hipMalloc((void**)&data.dev_outSrc, bitmap.image_size()));
    CUDA_CHECK_ERROR(hipMalloc((void**)&data.dev_constSrc, bitmap.image_size()));



    float *temp = new float[bitmap.image_size()];
    for (int i = 0; i < DIM * DIM; ++i) {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
            temp[i] = MAX_TEMP;
    }
    temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
    temp[DIM * 700 + 100] = MIN_TEMP;
    temp[DIM * 300 + 300] = MIN_TEMP;
    temp[DIM * 200 + 700] = MIN_TEMP;
    for (int y = 800; y < 900; ++y) {
        for (int x = 400; x < 500; ++x)
            temp[x + y * DIM] = MIN_TEMP;
    }

    CUDA_CHECK_ERROR(hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(),
                                hipMemcpyHostToDevice));
    for (int y = 800; y < DIM; ++y) {
        for (int x = 0; x < 200; ++x) {
            temp[x + y * DIM] = MAX_TEMP;
        }
    }
    CUDA_CHECK_ERROR(hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(),
                                hipMemcpyHostToDevice));

    delete [] temp;

    bitmap.anim_and_exit((void (*)(void *, int))AnimGPU, ((void(*)(void*))AnimExit));
}
